#include "hip/hip_runtime.h"
﻿// attention_test_fp32.cu
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cstdio>
#include <cmath>
#include <vector>
#include <algorithm>
#include "com_neocoretechs_cublas_Gemm.h"
// ---------- Error helpers ----------
#define CHECK_CUDA(x) do { hipError_t err = (x); if (err != hipSuccess) { \
  fprintf(stderr, "CUDA error %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); return -1; } } while(0)

#define CHECK_CUBLAS(x) do { hipblasStatus_t st = (x); if (st != HIPBLAS_STATUS_SUCCESS) { \
  fprintf(stderr, "cuBLAS error %d at %s:%d\n", (int)st, __FILE__, __LINE__); return -2; } } while(0)

// ---------- Row-wise softmax ----------
__global__ void row_softmax_fp32(const float* __restrict__ S, float* __restrict__ A,
    int rows, int cols, int ldS, int ldA) {
    int r = blockIdx.x * blockDim.x + threadIdx.x;
    if (r >= rows) return;

    const float* srow = S + r * ldS;
    float* arow = A + r * ldA;

    // 1) max
    float m = -1e30f;
    for (int c = 0; c < cols; ++c) m = fmaxf(m, srow[c]);

    // 2) exp and sum
    float sum = 0.f;
    for (int c = 0; c < cols; ++c) {
        float e = __expf(srow[c] - m);
        arow[c] = e;
        sum += e;
    }

    // 3) normalize
    float inv = 1.0f / sum;
    for (int c = 0; c < cols; ++c) arow[c] *= inv;
}

static int softmax_rows_fp32(const float* d_S, float* d_A, int rows, int cols, int ldS, int ldA, hipStream_t stream) {
    int threads = 128;
    int blocks = (rows + threads - 1) / threads;
    row_softmax_fp32 << <blocks, threads, 0, stream >> > (d_S, d_A, rows, cols, ldS, ldA);
    return (hipGetLastError() == hipSuccess) ? 0 : -3;
}

// ---------- CPU baseline (optional, small sizes) ----------
static void cpu_attention_fp32(const float* hQ, const float* hK, const float* hV,
    float* hO, int Tq, int Tk, int d) {
    std::vector<float> S(Tq * Tk), A(Tq * Tk);

    // S = (Q K^T) * 1/sqrt(d), row-major
    float alpha = 1.0f / std::sqrt((float)d);
    for (int i = 0; i < Tq; ++i) {
        for (int j = 0; j < Tk; ++j) {
            float acc = 0.f;
            for (int k = 0; k < d; ++k) acc += hQ[i * d + k] * hK[j * d + k];
            S[i * Tk + j] = acc * alpha;
        }
    }

    // row-wise softmax
    for (int i = 0; i < Tq; ++i) {
        float m = -1e30f;
        for (int j = 0; j < Tk; ++j) m = std::max(m, S[i * Tk + j]);
        float sum = 0.f;
        for (int j = 0; j < Tk; ++j) { float e = std::exp(S[i * Tk + j] - m); A[i * Tk + j] = e; sum += e; }
        float inv = 1.f / sum;
        for (int j = 0; j < Tk; ++j) A[i * Tk + j] *= inv;
    }

    // O = A V
    for (int i = 0; i < Tq; ++i) {
        for (int c = 0; c < d; ++c) {
            float acc = 0.f;
            for (int j = 0; j < Tk; ++j) acc += A[i * Tk + j] * hV[j * d + c];
            hO[i * d + c] = acc;
        }
    }
}

// ---------- Main test function ----------
extern "C"
int attention_test_fp32(hipblasHandle_t handle,
    // Sizes
    int Tq, int Tk, int d,
    // Device pointers (row-major)
    const float* d_Q, int ldQ,      // [Tq x d], ldQ = d
    const float* d_K, int ldK,      // [Tk x d], ldK = d
    const float* d_V, int ldV,      // [Tk x d], ldV = d
    float* d_O, int ldO,            // [Tq x d], ldO = d
    // Workspaces
    float* d_S, int ldS,            // [Tq x Tk], ldS = Tk
    float* d_A, int ldA,            // [Tq x Tk], ldA = Tk
    // Options
    int enable_tf32,                // 1 to enable TF32 (Ampere+)
    int do_cpu_check,               // 1 to validate for small sizes
    // Output timings (ms)
    float* out_ms_qkt,
    float* out_ms_softmax,
    float* out_ms_av
) {
    // Basic asserts
    if (Tq <= 0 || Tk <= 0 || d <= 0) { fprintf(stderr, "Invalid sizes\n"); return -10; }
    if (ldQ != d || ldK != d || ldV != d || ldO != d || ldS != Tk || ldA != Tk) {
        fprintf(stderr, "Leading dims mismatch (expect row-major)\n"); return -11;
    }
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));
    CHECK_CUBLAS(hipblasSetStream(handle, stream));
    if (enable_tf32) {
        // Best-effort: enable TF32 tensor ops mode (ignored on non-Ampere/Hopper)
        CHECK_CUBLAS(hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH));
    }
    // Events for timing
    hipEvent_t e0, e1, e2, e3, e4;
    CHECK_CUDA(hipEventCreate(&e0));
    CHECK_CUDA(hipEventCreate(&e1));
    CHECK_CUDA(hipEventCreate(&e2));
    CHECK_CUDA(hipEventCreate(&e3));
    CHECK_CUDA(hipEventCreate(&e4));

    // ---- QK^T scaled: S = (Q K^T) * alpha
    float alpha = 1.0f / std::sqrt((float)d);
    float beta = 0.0f;
    // m = Tq, n = Tk, k = d
    hipblasOperation_t opQ = HIPBLAS_OP_N;
    hipblasOperation_t opK = HIPBLAS_OP_T;

    CHECK_CUDA(hipEventRecord(e0, stream));
    CHECK_CUBLAS(hipblasSgemm(
        handle, opQ, opK,
        Tq, Tk, d,
        &alpha,
        d_Q, ldQ,
        d_K, ldK,
        &beta,
        d_S, ldS
    ));
    CHECK_CUDA(hipEventRecord(e1, stream));

    // ---- Softmax rows: A = softmax(S)
    int rc = softmax_rows_fp32(d_S, d_A, Tq, Tk, ldS, ldA, stream);
    if (rc) { fprintf(stderr, "Softmax kernel failed\n"); return rc; }
    CHECK_CUDA(hipEventRecord(e2, stream));

    // ---- AV: O = A V
    CHECK_CUBLAS(hipblasSgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        Tq, d, Tk,
        &alpha /* or 1.0f if you don't want extra scale */,
        d_A, ldA,
        d_V, ldV,
        &beta,
        d_O, ldO
    ));
    CHECK_CUDA(hipEventRecord(e3, stream));
    // ---- Sync and timings
    CHECK_CUDA(hipEventRecord(e4, stream));
    CHECK_CUDA(hipEventSynchronize(e4));
    float ms_qkt = 0, ms_softmax = 0, ms_av = 0;
    CHECK_CUDA(hipEventElapsedTime(&ms_qkt, e0, e1));
    CHECK_CUDA(hipEventElapsedTime(&ms_softmax, e1, e2));
    CHECK_CUDA(hipEventElapsedTime(&ms_av, e2, e3));
    if (out_ms_qkt) *out_ms_qkt = ms_qkt;
    if (out_ms_softmax) *out_ms_softmax = ms_softmax;
    if (out_ms_av) *out_ms_av = ms_av;

    // ---- Optional CPU baseline check (small sizes only)
    if (do_cpu_check && Tq <= 2 && Tk <= 256 && d <= 128) {
        std::vector<float> hQ(Tq * d), hK(Tk * d), hV(Tk * d), hO(Tq * d), hO_cpu(Tq * d);
        CHECK_CUDA(hipMemcpy(hQ.data(), d_Q, sizeof(float) * Tq * d, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(hK.data(), d_K, sizeof(float) * Tk * d, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(hV.data(), d_V, sizeof(float) * Tk * d, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(hO.data(), d_O, sizeof(float) * Tq * d, hipMemcpyDeviceToHost));
        cpu_attention_fp32(hQ.data(), hK.data(), hV.data(), hO_cpu.data(), Tq, Tk, d);
        // Compare a few elements
        int mismatches = 0;
        for (int i = 0; i < std::min(16, Tq * d); ++i) {
            float a = hO[i], b = hO_cpu[i];
            float diff = std::abs(a - b);
            if (diff > 1e-3f) ++mismatches;
        }
        if (mismatches) {
            fprintf(stderr, "CPU vs GPU mismatch count=%d (tolerance 1e-3)\n", mismatches);
            // Not failing hard; you can tighten if needed
        }
    }
    // Cleanup
    hipEventDestroy(e0); hipEventDestroy(e1); hipEventDestroy(e2); hipEventDestroy(e3); hipEventDestroy(e4);
    hipblasDestroy(handle);
    hipStreamDestroy(stream);

    // Report
    fprintf(stdout, "QK^T: %.3f ms | Softmax: %.3f ms | AV: %.3f ms | Tq=%d Tk=%d d=%d\n",
        ms_qkt, ms_softmax, ms_av, Tq, Tk, d);
    return 0;
}

JNIEXPORT jfloatArray JNICALL Java_com_neocoretechs_cublas_Attn_softMax
(JNIEnv* env, jclass clazz, jfloatArray jInput, jint rows, jint cols) {

    jsize len = env->GetArrayLength(jInput);
    std::vector<float> hInput(len);
    env->GetFloatArrayRegion(jInput, 0, len, hInput.data());

    std::vector<float> hOutput(len);

    float* dS, * dA;
    size_t bytes = len * sizeof(float);
    hipMalloc(&dS, bytes);
    hipMalloc(&dA, bytes);
    hipMemcpy(dS, hInput.data(), bytes, hipMemcpyHostToDevice);

    int threads = 128;
    int blocks = (rows + threads - 1) / threads;
    row_softmax_fp32 << <blocks, threads >> > (dS, dA, rows, cols, cols, cols);
    hipDeviceSynchronize();

    hipMemcpy(hOutput.data(), dA, bytes, hipMemcpyDeviceToHost);

    hipFree(dS);
    hipFree(dA);

    jfloatArray jOut = env->NewFloatArray(len);
    env->SetFloatArrayRegion(jOut, 0, len, hOutput.data());
    return jOut;
}

struct Ctx {
    hipblasHandle_t handle;
    int Tq, Tk, d;
    // optional: persistent device buffers
    float* dQ, * dK, * dV, * dO, * dS, * dA;
};

JNIEXPORT jlong JNICALL Java_com_neocoretechs_cublas_Attn_initContext(
    JNIEnv* env, jclass clazz, jlong cublasHandle, jint Tq, jint Tk, jint d, jint enableTf32) {
    auto* ctx = new Ctx{ (hipblasHandle_t)cublasHandle, Tq, Tk, d };
    // Allocate persistent device buffers once
    size_t bQ = size_t(Tq) * d * sizeof(float);
    size_t bK = size_t(Tk) * d * sizeof(float);
    size_t bV = size_t(Tk) * d * sizeof(float);
    size_t bO = size_t(Tq) * d * sizeof(float);
    size_t bS = size_t(Tq) * Tk * sizeof(float);
    size_t bA = size_t(Tq) * Tk * sizeof(float);
    hipMalloc(&ctx->dQ, bQ);
    hipMalloc(&ctx->dK, bK);
    hipMalloc(&ctx->dV, bV);
    hipMalloc(&ctx->dO, bO);
    hipMalloc(&ctx->dS, bS);
    hipMalloc(&ctx->dA, bA);
    // Optionally set TF32 math mode via cuBLAS, etc.
    return reinterpret_cast<jlong>(ctx);
}

JNIEXPORT void JNICALL Java_com_neocoretechs_cublas_Attn_freeContext(JNIEnv* env, jclass clazz, jlong h) {
    auto* ctx = reinterpret_cast<Ctx*>(h);
    if (!ctx) return;
    hipFree(ctx->dQ); hipFree(ctx->dK); hipFree(ctx->dV);
    hipFree(ctx->dO); hipFree(ctx->dS); hipFree(ctx->dA);
    delete ctx;
}

static float* addr(JNIEnv* env, jobject buf) {
    return static_cast<float*>(env->GetDirectBufferAddress(buf));
}

JNIEXPORT jint JNICALL Java_com_neocoretechs_cublas_Attn_attentionFp32(JNIEnv * env, jclass clazz, jlong h,
    jobject jQ, jint ldQ, jobject jK, jint ldK, jobject jV, jint ldV,
    jobject jO, jint ldO, jobject jS, jint ldS, jobject jA, jint ldA,
    jint doCpuCheck, jobject jMsQKT, jobject jMsSM, jobject jMsAV) {

    auto* ctx = reinterpret_cast<Ctx*>(h);
    float* hQ = addr(env, jQ);
    float* hK = addr(env, jK);
    float* hV = addr(env, jV);
    float* hO = addr(env, jO);
    float* hS = addr(env, jS);
    float* hA = addr(env, jA);

    float* msQKT = addr(env, jMsQKT);
    float* msSM = addr(env, jMsSM);
    float* msAV = addr(env, jMsAV);

    size_t bQ = size_t(ctx->Tq) * ctx->d * sizeof(float);
    size_t bK = size_t(ctx->Tk) * ctx->d * sizeof(float);
    size_t bV = size_t(ctx->Tk) * ctx->d * sizeof(float);
    size_t bO = size_t(ctx->Tq) * ctx->d * sizeof(float);
    size_t bS = size_t(ctx->Tq) * ctx->Tk * sizeof(float);
    size_t bA = size_t(ctx->Tq) * ctx->Tk * sizeof(float);

    // Host→Device (persistent device buffers)
    hipMemcpy(ctx->dQ, hQ, bQ, hipMemcpyHostToDevice);
    hipMemcpy(ctx->dK, hK, bK, hipMemcpyHostToDevice);
    hipMemcpy(ctx->dV, hV, bV, hipMemcpyHostToDevice);

    int ret = attention_test_fp32(ctx->handle, ctx->Tq, ctx->Tk, ctx->d,
        ctx->dQ, ldQ, ctx->dK, ldK, ctx->dV, ldV,
        ctx->dO, ldO, ctx->dS, ldS, ctx->dA, ldA,
        /*enable_tf32=*/1, doCpuCheck,
        msQKT, msSM, msAV);

    // Device→Host
    hipMemcpy(hO, ctx->dO, bO, hipMemcpyDeviceToHost);
    hipMemcpy(hS, ctx->dS, bS, hipMemcpyDeviceToHost);
    hipMemcpy(hA, ctx->dA, bA, hipMemcpyDeviceToHost);

    return ret;
}
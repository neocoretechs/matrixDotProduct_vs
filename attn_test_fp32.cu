#include "hip/hip_runtime.h"
// attention_test_fp32.cu
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cstdio>
#include <cmath>
#include <vector>
#include <algorithm>
#include "com_neocoretechs_cublas_Gemm.h"
// ---------- Error helpers ----------
#define CHECK_CUDA(x) do { hipError_t err = (x); if (err != hipSuccess) { \
  fprintf(stderr, "CUDA error %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__); return -1; } } while(0)

#define CHECK_CUBLAS(x) do { hipblasStatus_t st = (x); if (st != HIPBLAS_STATUS_SUCCESS) { \
  fprintf(stderr, "cuBLAS error %d at %s:%d\n", (int)st, __FILE__, __LINE__); return -2; } } while(0)

// ---------- Row-wise softmax ----------
__global__ void row_softmax_fp32(const float* __restrict__ S, float* __restrict__ A,
    int rows, int cols, int ldS, int ldA) {
    int r = blockIdx.x * blockDim.x + threadIdx.x;
    if (r >= rows) return;

    const float* srow = S + r * ldS;
    float* arow = A + r * ldA;

    // 1) max
    float m = -1e30f;
    for (int c = 0; c < cols; ++c) m = fmaxf(m, srow[c]);

    // 2) exp and sum
    float sum = 0.f;
    for (int c = 0; c < cols; ++c) {
        float e = __expf(srow[c] - m);
        arow[c] = e;
        sum += e;
    }

    // 3) normalize
    float inv = 1.0f / sum;
    for (int c = 0; c < cols; ++c) arow[c] *= inv;
}

static int softmax_rows_fp32(const float* d_S, float* d_A, int rows, int cols, int ldS, int ldA, hipStream_t stream) {
    int threads = 128;
    int blocks = (rows + threads - 1) / threads;
    row_softmax_fp32 << <blocks, threads, 0, stream >> > (d_S, d_A, rows, cols, ldS, ldA);
    return (hipGetLastError() == hipSuccess) ? 0 : -3;
}

// ---------- CPU baseline (optional, small sizes) ----------
static void cpu_attention_fp32(const float* hQ, const float* hK, const float* hV,
    float* hO, int Tq, int Tk, int d) {
    std::vector<float> S(Tq * Tk), A(Tq * Tk);

    // S = (Q K^T) * 1/sqrt(d), row-major
    float alpha = 1.0f / std::sqrt((float)d);
    for (int i = 0; i < Tq; ++i) {
        for (int j = 0; j < Tk; ++j) {
            float acc = 0.f;
            for (int k = 0; k < d; ++k) acc += hQ[i * d + k] * hK[j * d + k];
            S[i * Tk + j] = acc * alpha;
        }
    }

    // row-wise softmax
    for (int i = 0; i < Tq; ++i) {
        float m = -1e30f;
        for (int j = 0; j < Tk; ++j) m = std::max(m, S[i * Tk + j]);
        float sum = 0.f;
        for (int j = 0; j < Tk; ++j) { float e = std::exp(S[i * Tk + j] - m); A[i * Tk + j] = e; sum += e; }
        float inv = 1.f / sum;
        for (int j = 0; j < Tk; ++j) A[i * Tk + j] *= inv;
    }

    // O = A V
    for (int i = 0; i < Tq; ++i) {
        for (int c = 0; c < d; ++c) {
            float acc = 0.f;
            for (int j = 0; j < Tk; ++j) acc += A[i * Tk + j] * hV[j * d + c];
            hO[i * d + c] = acc;
        }
    }
}

// ---------- Main test function ----------
extern "C"
int attention_test_fp32(
    // Sizes
    int Tq, int Tk, int d,
    // Device pointers (row-major)
    const float* d_Q, int ldQ,      // [Tq x d], ldQ = d
    const float* d_K, int ldK,      // [Tk x d], ldK = d
    const float* d_V, int ldV,      // [Tk x d], ldV = d
    float* d_O, int ldO,            // [Tq x d], ldO = d
    // Workspaces
    float* d_S, int ldS,            // [Tq x Tk], ldS = Tk
    float* d_A, int ldA,            // [Tq x Tk], ldA = Tk
    // Options
    int enable_tf32,                // 1 to enable TF32 (Ampere+)
    int do_cpu_check,               // 1 to validate for small sizes
    // Output timings (ms)
    float* out_ms_qkt,
    float* out_ms_softmax,
    float* out_ms_av
) {
    // Basic asserts
    if (Tq <= 0 || Tk <= 0 || d <= 0) { fprintf(stderr, "Invalid sizes\n"); return -10; }
    if (ldQ != d || ldK != d || ldV != d || ldO != d || ldS != Tk || ldA != Tk) {
        fprintf(stderr, "Leading dims mismatch (expect row-major)\n"); return -11;
    }

    // Create cuBLAS handle and stream
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));
    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));
    CHECK_CUBLAS(hipblasSetStream(handle, stream));
    if (enable_tf32) {
        // Best-effort: enable TF32 tensor ops mode (ignored on non-Ampere/Hopper)
        CHECK_CUBLAS(hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH));
    }

    // Events for timing
    hipEvent_t e0, e1, e2, e3, e4;
    CHECK_CUDA(hipEventCreate(&e0));
    CHECK_CUDA(hipEventCreate(&e1));
    CHECK_CUDA(hipEventCreate(&e2));
    CHECK_CUDA(hipEventCreate(&e3));
    CHECK_CUDA(hipEventCreate(&e4));

    // ---- QK^T scaled: S = (Q K^T) * alpha
    float alpha = 1.0f / std::sqrt((float)d);
    float beta = 0.0f;
    // m = Tq, n = Tk, k = d
    hipblasOperation_t opQ = HIPBLAS_OP_N;
    hipblasOperation_t opK = HIPBLAS_OP_T;

    CHECK_CUDA(hipEventRecord(e0, stream));
    CHECK_CUBLAS(hipblasSgemm(
        handle, opQ, opK,
        Tq, Tk, d,
        &alpha,
        d_Q, ldQ,
        d_K, ldK,
        &beta,
        d_S, ldS
    ));
    CHECK_CUDA(hipEventRecord(e1, stream));

    // ---- Softmax rows: A = softmax(S)
    int rc = softmax_rows_fp32(d_S, d_A, Tq, Tk, ldS, ldA, stream);
    if (rc) { fprintf(stderr, "Softmax kernel failed\n"); return rc; }
    CHECK_CUDA(hipEventRecord(e2, stream));

    // ---- AV: O = A V
    CHECK_CUBLAS(hipblasSgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        Tq, d, Tk,
        &alpha /* or 1.0f if you don't want extra scale */,
        d_A, ldA,
        d_V, ldV,
        &beta,
        d_O, ldO
    ));
    CHECK_CUDA(hipEventRecord(e3, stream));

    // ---- Sync and timings
    CHECK_CUDA(hipEventRecord(e4, stream));
    CHECK_CUDA(hipEventSynchronize(e4));
    float ms_qkt = 0, ms_softmax = 0, ms_av = 0;
    CHECK_CUDA(hipEventElapsedTime(&ms_qkt, e0, e1));
    CHECK_CUDA(hipEventElapsedTime(&ms_softmax, e1, e2));
    CHECK_CUDA(hipEventElapsedTime(&ms_av, e2, e3));
    if (out_ms_qkt) *out_ms_qkt = ms_qkt;
    if (out_ms_softmax) *out_ms_softmax = ms_softmax;
    if (out_ms_av) *out_ms_av = ms_av;

    // ---- Optional CPU baseline check (small sizes only)
    if (do_cpu_check && Tq <= 2 && Tk <= 256 && d <= 128) {
        std::vector<float> hQ(Tq * d), hK(Tk * d), hV(Tk * d), hO(Tq * d), hO_cpu(Tq * d);
        CHECK_CUDA(hipMemcpy(hQ.data(), d_Q, sizeof(float) * Tq * d, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(hK.data(), d_K, sizeof(float) * Tk * d, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(hV.data(), d_V, sizeof(float) * Tk * d, hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(hO.data(), d_O, sizeof(float) * Tq * d, hipMemcpyDeviceToHost));
        cpu_attention_fp32(hQ.data(), hK.data(), hV.data(), hO_cpu.data(), Tq, Tk, d);
        // Compare a few elements
        int mismatches = 0;
        for (int i = 0; i < std::min(16, Tq * d); ++i) {
            float a = hO[i], b = hO_cpu[i];
            float diff = std::abs(a - b);
            if (diff > 1e-3f) ++mismatches;
        }
        if (mismatches) {
            fprintf(stderr, "CPU vs GPU mismatch count=%d (tolerance 1e-3)\n", mismatches);
            // Not failing hard; you can tighten if needed
        }
    }

    // Cleanup
    hipEventDestroy(e0); hipEventDestroy(e1); hipEventDestroy(e2); hipEventDestroy(e3); hipEventDestroy(e4);
    hipblasDestroy(handle);
    hipStreamDestroy(stream);

    // Report
    fprintf(stdout, "QK^T: %.3f ms | Softmax: %.3f ms | AV: %.3f ms | Tq=%d Tk=%d d=%d\n",
        ms_qkt, ms_softmax, ms_av, Tq, Tk, d);

    return 0;
}

JNIEXPORT jfloatArray JNICALL Java_com_neocoretechs_cublas_Attn_softMax
(JNIEnv* env, jclass clazz, jfloatArray jInput, jint rows, jint cols) {

    jsize len = env->GetArrayLength(jInput);
    std::vector<float> hInput(len);
    env->GetFloatArrayRegion(jInput, 0, len, hInput.data());

    std::vector<float> hOutput(len);

    float* dS, * dA;
    size_t bytes = len * sizeof(float);
    hipMalloc(&dS, bytes);
    hipMalloc(&dA, bytes);
    hipMemcpy(dS, hInput.data(), bytes, hipMemcpyHostToDevice);

    int threads = 128;
    int blocks = (rows + threads - 1) / threads;
    row_softmax_fp32 << <blocks, threads >> > (dS, dA, rows, cols, cols, cols);
    hipDeviceSynchronize();

    hipMemcpy(hOutput.data(), dA, bytes, hipMemcpyDeviceToHost);

    hipFree(dS);
    hipFree(dA);

    jfloatArray jOut = env->NewFloatArray(len);
    env->SetFloatArrayRegion(jOut, 0, len, hOutput.data());
    return jOut;
}